#include "hip/hip_runtime.h"
extern "C++"

#include "GPUJRand.h"
#include "GPUStructs.h"
#include <math.h>

__device__ inline int filterEnchantments(bool* incompatible, AvailableEnchantmentResult* availableEnchantmentResultPtr, int enchantmentId, int enchantmentMatrixSize, bool** enchantmentMatrix) {
    AvailableEnchantmentResult availableEnchantmentResult = *availableEnchantmentResultPtr;
    int emptyCount = 0;
    for (char i = 0; i < availableEnchantmentResult.length; i++) {
        if (incompatible[i]) {
            emptyCount++;
        } else {
           if (!enchantmentMatrix[enchantmentId][availableEnchantmentResult.availableEnchantments[i]]) {
               incompatible[i] = true;
               emptyCount++;
           }
        }
    }
    return emptyCount;
}

extern "C" __global__ void run(long long amountInBatch, long long batchNr, int seedsPerCudaJob, long long startingSeed, int amountOfLootPools, LootPool lootPools[], int requirementsSize, Item requirements[], int enchantmentMatrixSize, bool** enchantmentMatrix, long long *output)
{

    int indexInBatch = blockIdx.x * blockDim.x + threadIdx.x;
    if (indexInBatch < amountInBatch / seedsPerCudaJob) {

        int64_t firstSeedInBatch = batchNr * amountInBatch;
        int longsPerCudaJob = seedsPerCudaJob / 64;
        int64_t firstLongForJob = (firstSeedInBatch / 64) + indexInBatch * longsPerCudaJob;

        for (int p = 0; p < longsPerCudaJob; p++) {
            int indexInOutput = indexInBatch * longsPerCudaJob + p;
            long long outputLong = 0;
            int64_t currentLong = (firstLongForJob + p);

            for (int indexInLong = 0; indexInLong < 64; indexInLong++) {
                int64_t seed = 0;
                int64_t *seedPtr = &seed;

                int64_t lootSeed = 64 * currentLong + indexInLong + startingSeed;

                setSeedFromInt(seedPtr, lootSeed);

                int requirementCounts[10];

                for (int i = 0; i < requirementsSize; i++) {
                    requirementCounts[i] = requirements[i].count;
                }

                for (int i = 0; i < amountOfLootPools; i++) {
                    LootPool lootPool = lootPools[i];

                    bool singlePool = true;
                    int firstItemId = lootPool.optimizationArray[0].id;
                    for (int item = 0; item < lootPool.optimizationArrayLength; item++){
                        if (lootPool.optimizationArray[item].id!=firstItemId) {
                            singlePool = false;
                        }
                    }
                    int rolls = 0;
                    int difference = (lootPool.max - lootPool.min + 1);
                    if (lootPool.max != lootPool.min) {
                        nextInt(seedPtr, difference, &rolls);
                    }

                    rolls = rolls + lootPool.min;
                    for (int roll = 0; roll < rolls; roll++) {
                        int index = 0;

                        ItemRoll itemRoll;
                        if (!singlePool) {
                            nextInt(seedPtr, lootPool.optimizationArrayLength, &index);
                            itemRoll = lootPool.optimizationArray[index];
                        } else {
                            itemRoll = lootPool.optimizationArray[0];
                        }

//                        printf("%i, %i, %i, %i, %i, %i, %i, %i, %i, %i\n", itemRoll.id, itemRoll.min, itemRoll.max, itemRoll.enchantability, itemRoll.minEnchantLevel, itemRoll.maxEnchantLevel, itemRoll.padding, itemRoll.enchantRandomlyFunction, itemRoll.applicableEnchantmentsLength);

                        // id 0 is empty roll.
                        if (itemRoll.id == 0) {
                            continue;
                        }

                        Item item;
                        item.id = itemRoll.id;
                        item.enchantmentCount = 0;

                        int enchantmentIds[10];
                        int enchantmentLevels[10];

                        item.enchantmentIds = enchantmentIds;
                        item.enchantmentLevels = enchantmentLevels;

                        if (itemRoll.max == itemRoll.min) {
                            item.count = itemRoll.max;
                        } else {
                            int number = 0;
                            nextInt(seedPtr, itemRoll.max - itemRoll.min + 1, &number);
                            item.count = itemRoll.min + number;
                        }

                        // Function ordering:
                        // ApplyDamageFunction -> EnchantRandomlyFunction

                        if (itemRoll.applyDamageFunction) {
                            advance(seedPtr);
                        }

                        if (itemRoll.enchantRandomlyFunction) {
                            int enchantmentIndex = 0;
//                            printf("ENCHANT RANDOMLY \n");

                            nextInt(seedPtr, itemRoll.applicableEnchantmentsLength, &enchantmentIndex);
                            int enchantmentId = itemRoll.applicableEnchantments[enchantmentIndex];
                            item.enchantmentIds[0] = enchantmentId;
                            if (enchantmentId > 9) {
                                int enchantmentLevel = 0;
                                nextInt(seedPtr, itemRoll.enchantmentLevels[enchantmentIndex], &enchantmentLevel);
                                item.enchantmentLevels[0] = enchantmentLevel + 1;
                            } else {
                                item.enchantmentLevels[0] = 1;
                            }
                            item.enchantmentCount++;

                        }

                        if (itemRoll.enchantWithLevelFunction) {
                            bool incompatible[152];

                            #pragma unroll
                            for (int o = 0; o < 152; o++) {
                                incompatible[o] = false;
                            }

                            int level = 0;
                            if (itemRoll.maxEnchantLevel == itemRoll.minEnchantLevel) {
                                level = itemRoll.minEnchantLevel;
                            } else {
                                nextInt(seedPtr, itemRoll.maxEnchantLevel - itemRoll.minEnchantLevel + 1, &level);
                                level += itemRoll.minEnchantLevel;
                            }

                            level += 1;

                            int randomCall1 = 0;
                            int randomCall2 = 0;

                            nextInt(seedPtr, itemRoll.enchantability / 4 + 1, &randomCall1);
                            nextInt(seedPtr, itemRoll.enchantability / 4 + 1, &randomCall2);

                            level += randomCall1 + randomCall2;

                            float amplifier = 0;

                            amplifier = (nextFloat(seedPtr) + nextFloat(seedPtr) - 1) * 0.15;

                            level = round(level + level * amplifier);

                            if (level < 1) {
                                level = 1;
                            }

                            AvailableEnchantmentResult availableEnchantments = itemRoll.availableEnchantmentResults[level];
                            if (availableEnchantments.length > 0) {
                                int index = 0;
                                nextInt(seedPtr, availableEnchantments.length, &index);

                                int enchantmentId = availableEnchantments.availableEnchantments[index];
                                int enchantmentLevel = availableEnchantments.availableEnchantmentLevels[index];

                                item.enchantmentIds[item.enchantmentCount] = enchantmentId;
                                item.enchantmentLevels[item.enchantmentCount] = enchantmentLevel;
                                item.enchantmentCount++;

                                int whileNextInt = 0;
                                nextInt(seedPtr, 50, &whileNextInt);


                                while (whileNextInt <= level) {
                                    int emptyCount = filterEnchantments(incompatible, &availableEnchantments, enchantmentId, enchantmentMatrixSize, enchantmentMatrix);
                                    if (emptyCount == availableEnchantments.length) {
                                        break;
                                    }

                                    index = 0;
                                    nextInt(seedPtr, availableEnchantments.length - emptyCount, &index);
                                    for (int j = 0; j < availableEnchantments.length; j++) {
                                        if(!incompatible[j]){
                                           index--;
                                        }

                                        if (index < 0) {
                                            enchantmentId = availableEnchantments.availableEnchantments[j];
                                            enchantmentLevel = availableEnchantments.availableEnchantmentLevels[j];
                                            break;
                                        }
                                    }

                                    item.enchantmentIds[item.enchantmentCount] = enchantmentId;
                                    item.enchantmentLevels[item.enchantmentCount] = enchantmentLevel;
                                    item.enchantmentCount++;

                                    level /= 2;
                                    nextInt(seedPtr, 50, &whileNextInt);
                                }
                            }
                        }

                        if (itemRoll.effectFunction) {
                            int effectId = 0;
                            nextInt(seedPtr, 6, &effectId);
                            advance(seedPtr);
                            //TODO: Store effect id with item.
                        }

//                        printf("%i, %i, %i \n", item.id, item.count, item.enchantmentCount);

                        for (int requirementIndex = 0; requirementIndex<requirementsSize; requirementIndex++){
                            Item requirement = requirements[requirementIndex];
                            if (item.id == requirement.id) {
                            if (requirement.enchantmentCount == item.enchantmentCount) {
                                    bool matches = true;
                                    for (int enchIdx = 0; enchIdx < requirement.enchantmentCount; enchIdx++) {
                                        bool found = false;
                                        for (int enchIdx2 = 0; enchIdx2 < item.enchantmentCount; enchIdx2++) {
                                            if (requirement.enchantmentIds[enchIdx] == item.enchantmentIds[enchIdx2]) {
                                                if (requirement.enchantmentLevels[enchIdx] <= item.enchantmentLevels[enchIdx2]) {
                                                    found = true;
                                                    break;
                                                }
                                            }
                                        }

                                        if (!found) {
                                            matches = false;
                                            break;
                                        }
                                    }

                                    if (matches) {
                                        requirementCounts[requirementIndex] -= item.count;
                                    }
                                }
                            }
                        }
                    }
                }

                bool failed = false;
                for (int requirementIndex = 0; requirementIndex<requirementsSize; requirementIndex++){
                    if (requirementCounts[requirementIndex] > 0) {
                        failed = true;
                    }
                }
                if (!failed) {
                   outputLong += 1;
                }

                if (indexInLong != 63) {
                    outputLong = outputLong << 1;
                }
            }
            output[indexInOutput] = outputLong;
        }
    }
}